#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>
#include <iostream>

#include <ATen/ATen.h>
#include <ATen/NativeFunctions.h>
#include <ATen/Parallel.h>

// #include "utils.cuh"

#include <Eigen/Sparse>
#include <Eigen/SparseCore>
#include <Eigen/SparseCholesky>

typedef Eigen::SparseMatrix<double> SpMat;
typedef Eigen::Triplet<double> T;
typedef std::vector<std::vector<long>> graph_t;
typedef std::vector<torch::Tensor> tensor_list_t;



#define MIN_DEPTH 0.25
#define CV_PI 3.1415926

#define THREADS 256
#define NUM_BLOCKS(batch_size) ((batch_size + THREADS - 1) / THREADS)


#define GPU_1D_KERNEL_LOOP(k, n) \
  for (size_t k = threadIdx.x; k<n; k += blockDim.x)


__device__ void warpReduce(volatile float *sdata, unsigned int tid) {
  sdata[tid] += sdata[tid + 32];
  sdata[tid] += sdata[tid + 16];
  sdata[tid] += sdata[tid +  8];
  sdata[tid] += sdata[tid +  4];
  sdata[tid] += sdata[tid +  2];
  sdata[tid] += sdata[tid +  1];
}

__device__ void blockReduce(volatile float *sdata) {
  unsigned int tid = threadIdx.x;
  __syncthreads();

  // if (threadIdx.x < 256) {sdata[tid] += sdata[tid + 256]; } __syncthreads();
  if (threadIdx.x < 128) {sdata[tid] += sdata[tid + 128]; } __syncthreads();
  if (threadIdx.x <  64) {sdata[tid] += sdata[tid +  64]; } __syncthreads();

  if (tid < 32) warpReduce(sdata, tid);
  __syncthreads();
}

__device__ float2 proj(const float *Xj, const float *intrinsics)
{
  float2 xnyn;
  xnyn.x = intrinsics[0] * (Xj[0] / Xj[2]) + intrinsics[2];
  xnyn.y = intrinsics[1] * (Xj[1] / Xj[2]) + intrinsics[3];
  return xnyn;
}

__device__ void iproj(float u, float v, const float *intrinsics, float * X, float di)
{
  X[0] = (u - intrinsics[2]) / intrinsics[0];
  X[1] = (v - intrinsics[3]) / intrinsics[1];
  X[2] = 1;
  X[3] = di;
}

__device__ void
actSO3(const float *q, const float *X, float *Y) {
  float uv[3];
  uv[0] = 2.0 * (q[1]*X[2] - q[2]*X[1]);
  uv[1] = 2.0 * (q[2]*X[0] - q[0]*X[2]);
  uv[2] = 2.0 * (q[0]*X[1] - q[1]*X[0]);

  Y[0] = X[0] + q[3]*uv[0] + (q[1]*uv[2] - q[2]*uv[1]);
  Y[1] = X[1] + q[3]*uv[1] + (q[2]*uv[0] - q[0]*uv[2]);
  Y[2] = X[2] + q[3]*uv[2] + (q[0]*uv[1] - q[1]*uv[0]);
}

__device__  void
actSE3(const float *t, const float *q, const float *X, float *Y) {
  actSO3(q, X, Y);
  Y[3] = X[3];
  Y[0] += X[3] * t[0];
  Y[1] += X[3] * t[1];
  Y[2] += X[3] * t[2];
}

__device__ void
adjSE3(const float *t, const float *q, const float *X, float *Y) {
  float qinv[4] = {-q[0], -q[1], -q[2], q[3]};
  actSO3(qinv, &X[0], &Y[0]);
  actSO3(qinv, &X[3], &Y[3]);

  float u[3], v[3];
  u[0] = t[2]*X[1] - t[1]*X[2];
  u[1] = t[0]*X[2] - t[2]*X[0];
  u[2] = t[1]*X[0] - t[0]*X[1];

  actSO3(qinv, u, v);
  Y[3] += v[0];
  Y[4] += v[1];
  Y[5] += v[2];
}

__device__ void 
relSE3(const float *ti, const float *qi, const float *tj, const float *qj, float *tij, float *qij) {
  qij[0] = -qj[3] * qi[0] + qj[0] * qi[3] - qj[1] * qi[2] + qj[2] * qi[1],
  qij[1] = -qj[3] * qi[1] + qj[1] * qi[3] - qj[2] * qi[0] + qj[0] * qi[2],
  qij[2] = -qj[3] * qi[2] + qj[2] * qi[3] - qj[0] * qi[1] + qj[1] * qi[0],
  qij[3] =  qj[3] * qi[3] + qj[0] * qi[0] + qj[1] * qi[1] + qj[2] * qi[2],

  actSO3(qij, ti, tij);
  tij[0] = tj[0] - tij[0];
  tij[1] = tj[1] - tij[1];
  tij[2] = tj[2] - tij[2];
}

  
__device__ void
expSO3(const float *phi, float* q) {
  // SO3 exponential map
  float theta_sq = phi[0]*phi[0] + phi[1]*phi[1] + phi[2]*phi[2];
  float theta_p4 = theta_sq * theta_sq;

  float theta = sqrtf(theta_sq);
  float imag, real;

  if (theta_sq < 1e-8) {
    imag = 0.5 - (1.0/48.0)*theta_sq + (1.0/3840.0)*theta_p4;
    real = 1.0 - (1.0/ 8.0)*theta_sq + (1.0/ 384.0)*theta_p4;
  } else {
    imag = sinf(0.5 * theta) / theta;
    real = cosf(0.5 * theta);
  }

  q[0] = imag * phi[0];
  q[1] = imag * phi[1];
  q[2] = imag * phi[2];
  q[3] = real;

}

__device__ void
crossInplace(const float* a, float *b) {
  float x[3] = {
    a[1]*b[2] - a[2]*b[1],
    a[2]*b[0] - a[0]*b[2],
    a[0]*b[1] - a[1]*b[0], 
  };

  b[0] = x[0];
  b[1] = x[1];
  b[2] = x[2];
}

__device__ void
expSE3(const float *xi, float* t, float* q) {
  // SE3 exponential map

  expSO3(xi + 3, q);
  float tau[3] = {xi[0], xi[1], xi[2]};
  float phi[3] = {xi[3], xi[4], xi[5]};

  float theta_sq = phi[0]*phi[0] + phi[1]*phi[1] + phi[2]*phi[2];
  float theta = sqrtf(theta_sq);

  t[0] = tau[0]; 
  t[1] = tau[1]; 
  t[2] = tau[2];

  if (theta > 1e-4) {
    float a = (1 - cosf(theta)) / theta_sq;
    crossInplace(phi, tau);
    t[0] += a * tau[0];
    t[1] += a * tau[1];
    t[2] += a * tau[2];

    float b = (theta - sinf(theta)) / (theta * theta_sq);
    crossInplace(phi, tau);
    t[0] += b * tau[0];
    t[1] += b * tau[1];
    t[2] += b * tau[2];
  }
}

__global__ void projective_transform2_kernel(
    const torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> target,
    const torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> weight,
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> poses,
    const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> disps,
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> intrinsics,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> ii,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> jj,
    torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> Cii,
    torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> bz)
{
  const int block_id = blockIdx.x;
  const int thread_id = threadIdx.x;

  const int ht = disps.size(1);
  const int wd = disps.size(2);

  int ix = static_cast<int>(ii[block_id]);
  int jx = static_cast<int>(jj[block_id]);

  __shared__ float intrinsics_[4];
  __shared__ float fx;
  __shared__ float fy;

  __shared__ float ti[3], tj[3], tij[3];
  __shared__ float qi[4], qj[4], qij[4];

  // load intrinsics from global memory
  if (thread_id == 0) {
    intrinsics_[0] = intrinsics[0][0];
    intrinsics_[1] = intrinsics[0][1];
    intrinsics_[2] = intrinsics[0][2];
    intrinsics_[3] = intrinsics[0][3];
    fx = intrinsics[0][0];
    fy = intrinsics[0][1];
  }

  __syncthreads();

  // load poses from global memory
  if (thread_id < 3) {
    ti[thread_id] = poses[ix][thread_id];
    tj[thread_id] = poses[jx][thread_id];
  }

  if (thread_id < 4) {
    qi[thread_id] = poses[ix][thread_id+3];
    qj[thread_id] = poses[jx][thread_id+3];
  }

  __syncthreads();

  if (thread_id == 0) {
    relSE3(ti, qi, tj, qj, tij, qij);
  }

  __syncthreads();

  //points 
  float Xi[4];
  float Xj[4];
  float2 xnyn;

  // jacobians
  float Jp[6];
  float Jz;

  __syncthreads();

  GPU_1D_KERNEL_LOOP(k, ht*wd) {

    const int i = k / wd;
    const int j = k % wd;

    const float u = static_cast<float>(j);
    const float v = static_cast<float>(i);
    
    // homogenous coordinates
    iproj(u, v, intrinsics_, Xi, disps[ix][i][j]);

    // transform homogenous point
    actSE3(tij, qij, Xi, Xj);

    xnyn = proj(Xj, intrinsics_);

    const float x = Xj[0];
    const float y = Xj[1];
    const float z = (Xj[2] < MIN_DEPTH) ? 0.0 : Xj[2];
    const float h = Xj[3];

    const float d = (Xj[2] < MIN_DEPTH) ? 0.0 : 1.0 / Xj[2];
    const float d2 = d * d;

    float wu = (Xj[2] < MIN_DEPTH) ? 0.0 : .001 * weight[block_id][0][i][j];
    float wv = (Xj[2] < MIN_DEPTH) ? 0.0 : .001 * weight[block_id][1][i][j];
    const float ru = target[block_id][0][i][j] - xnyn.x;
    const float rv = target[block_id][1][i][j] - xnyn.y;

    // assume pinhole
    Jp[0] = fx * d;
    Jp[1] = 0;
    Jp[2] = fx * (-x * d2);
    Jp[3] = 0;
    Jp[4] = fy * d;
    Jp[5] = fy * (-y * d2);

    // x - coordinate
    Jz = Jp[0] * tij[0] + Jp[1] * tij[1] + Jp[2] * tij[2];
    Cii[block_id][k] = wu * Jz * Jz;
    bz[block_id][k] = wu * ru * Jz;

    // y - coordinate
    Jz = Jp[3] * tij[0] + Jp[4] * tij[1] + Jp[5] * tij[2];
    Cii[block_id][k] += wv * Jz * Jz;
    bz[block_id][k] += wv * rv * Jz;
  }
}

__global__ void projective_transform_kernel(
    const torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> target,
    const torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> weight,
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> poses,
    const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> disps,
    const torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> intrinsics,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> ii,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> jj,
    torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> Hs,
    torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> vs,
    torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> Eii,
    torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> Eij,
    torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> Cii,
    torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> bz)
{
  const int block_id = blockIdx.x;
  const int thread_id = threadIdx.x;

  const int ht = disps.size(1);
  const int wd = disps.size(2);

  int ix = static_cast<int>(ii[block_id]);
  int jx = static_cast<int>(jj[block_id]);

  __shared__ float intrinsics_[4];
  __shared__ float fx;
  __shared__ float fy;

  __shared__ float ti[3], tj[3], tij[3];
  __shared__ float qi[4], qj[4], qij[4];

  // load intrinsics from global memory
  if (thread_id == 0) {
    intrinsics_[0] = intrinsics[0];
    intrinsics_[1] = intrinsics[1];
    intrinsics_[2] = intrinsics[2];
    intrinsics_[3] = intrinsics[3];
    fx = intrinsics[0];
    fy = intrinsics[1];
  }

  __syncthreads();

  // load poses from global memory
  if (thread_id < 3) {
    ti[thread_id] = poses[ix][thread_id];
    tj[thread_id] = poses[jx][thread_id];
  }

  if (thread_id < 4) {
    qi[thread_id] = poses[ix][thread_id+3];
    qj[thread_id] = poses[jx][thread_id+3];
  }

  __syncthreads();

  if (thread_id == 0) {
    relSE3(ti, qi, tj, qj, tij, qij);
  }

  __syncthreads();

  //points 
  float Xi[4];
  float Xj[4];
  float2 xnyn;

  // jacobians
  float Jp[6];
  float Jx[12];
  float Jz;

  float* Ji = &Jx[0];
  float* Jj = &Jx[6];

  // hessians
  float hij[12*(12+1)/2];

  float vi[6], vj[6];

  int l;
  for (l=0; l<12*(12+1)/2; l++) {
    hij[l] = 0;
  }

  for (int n=0; n<6; n++) {
    vi[n] = 0;
    vj[n] = 0;
  }

  __syncthreads();

  GPU_1D_KERNEL_LOOP(k, ht*wd) {

    const int i = k / wd;
    const int j = k % wd;

    const float u = static_cast<float>(j);
    const float v = static_cast<float>(i);
    
    // homogenous coordinates
    iproj(u, v, intrinsics_, Xi, disps[ix][i][j]);

    // transform homogenous point
    actSE3(tij, qij, Xi, Xj);

    xnyn = proj(Xj, intrinsics_);

    const float x = Xj[0];
    const float y = Xj[1];
    const float z = (Xj[2] < MIN_DEPTH) ? 0.0 : Xj[2];
    const float h = Xj[3];

    const float d = (Xj[2] < MIN_DEPTH) ? 0.0 : 1.0 / Xj[2];
    const float d2 = d * d;

    float wu = (Xj[2] < MIN_DEPTH) ? 0.0 : .001 * weight[block_id][0][i][j];
    float wv = (Xj[2] < MIN_DEPTH) ? 0.0 : .001 * weight[block_id][1][i][j];
    const float ru = target[block_id][0][i][j] - xnyn.x;
    const float rv = target[block_id][1][i][j] - xnyn.y;

    // assume pinhole
    Jp[0] = fx * d;
    Jp[1] = 0;
    Jp[2] = fx * (-x * d2);
    Jp[3] = 0;
    Jp[4] = fy * d;
    Jp[5] = fy * (-y * d2);

    // x - coordinate
    Jj[0] = Jp[0] * h;
    Jj[1] = Jp[1] * h;
    Jj[2] = Jp[2] * h;
    Jj[3] = -Jp[1] * z + Jp[2] * y;
    Jj[4] =  Jp[0] * z - Jp[2] * x;
    Jj[5] = -Jp[0] * y + Jp[1] * x;

    Jz = Jp[0] * tij[0] + Jp[1] * tij[1] + Jp[2] * tij[2];
    Cii[block_id][k] = wu * Jz * Jz;
    bz[block_id][k] = wu * ru * Jz;

    if (ix == jx) wu = 0;

    adjSE3(tij, qij, Jj, Ji);
    for (int n=0; n<6; n++) Ji[n] *= -1;

    l=0;
    for (int n=0; n<12; n++) {
      for (int m=0; m<=n; m++) {
        hij[l] += wu * Jx[n] * Jx[m];
        l++;
      }
    }

    for (int n=0; n<6; n++) {
      vi[n] += wu * ru * Ji[n];
      vj[n] += wu * ru * Jj[n];

      Eii[block_id][n][k] = wu * Jz * Ji[n];
      Eij[block_id][n][k] = wu * Jz * Jj[n];
    }

    // y - coordinate
    Jj[0] = Jp[3] * h;
    Jj[1] = Jp[4] * h;
    Jj[2] = Jp[5] * h;
    Jj[3] = -Jp[4] * z + Jp[5] * y;
    Jj[4] =  Jp[3] * z - Jp[5] * x;
    Jj[5] = -Jp[3] * y + Jp[4] * x;

    Jz = Jp[3] * tij[0] + Jp[4] * tij[1] + Jp[5] * tij[2];
    Cii[block_id][k] += wv * Jz * Jz;
    bz[block_id][k] += wv * rv * Jz;

    if (ix == jx) wv = 0;

    adjSE3(tij, qij, Jj, Ji);
    for (int n=0; n<6; n++) Ji[n] *= -1;

    l=0;
    for (int n=0; n<12; n++) {
      for (int m=0; m<=n; m++) {
        hij[l] += wv * Jx[n] * Jx[m];
        l++;
      }
    }

    for (int n=0; n<6; n++) {
      vi[n] += wv * rv * Ji[n];
      vj[n] += wv * rv * Jj[n];

      Eii[block_id][n][k] += wv * Jz * Ji[n];
      Eij[block_id][n][k] += wv * Jz * Jj[n];
    }


  }

  __syncthreads();

  __shared__ float sdata[THREADS];
  for (int n=0; n<6; n++) {
    sdata[threadIdx.x] = vi[n];
    blockReduce(sdata);
    if (threadIdx.x == 0) {
      vs[0][block_id][n] = sdata[0];
    }

    __syncthreads();

    sdata[threadIdx.x] = vj[n];
    blockReduce(sdata);
    if (threadIdx.x == 0) {
      vs[1][block_id][n] = sdata[0];
    }

  }

  l=0;
  for (int n=0; n<12; n++) {
    for (int m=0; m<=n; m++) {
      sdata[threadIdx.x] = hij[l];
      blockReduce(sdata);

      if (threadIdx.x == 0) {
        if (n<6 && m<6) {
          Hs[0][block_id][n][m] = sdata[0];
          Hs[0][block_id][m][n] = sdata[0];
        }
        else if (n >=6 && m<6) {
          Hs[1][block_id][m][n-6] = sdata[0];
          Hs[2][block_id][n-6][m] = sdata[0];
        }
        else {
          Hs[3][block_id][n-6][m-6] = sdata[0];
          Hs[3][block_id][m-6][n-6] = sdata[0];
        }
      }

      l++;
    }
  }
}


__global__ void frame_distance_kernel(
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> poses,
    const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> disps,
    const torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> intrinsics,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> ii,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> jj,
    torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> dist,
    const float beta) {

  const int block_id = blockIdx.x;
  const int thread_id = threadIdx.x;

  const int ht = disps.size(1);
  const int wd = disps.size(2);

  __shared__ int ix;
  __shared__ int jx;

  __shared__ float intrinsics_[4];

  __shared__ float ti[3], tj[3], tij[3];
  __shared__ float qi[4], qj[4], qij[4];

  // load intrinsics from global memory
  if (thread_id == 0) {
    ix = static_cast<int>(ii[block_id]);
    jx = static_cast<int>(jj[block_id]);
    intrinsics_[0] = intrinsics[0];
    intrinsics_[1] = intrinsics[1];
    intrinsics_[2] = intrinsics[2];
    intrinsics_[3] = intrinsics[3];
  }

  __syncthreads();


  //points 
  float Xi[4];
  float Xj[4];

  __shared__ float accum[THREADS]; accum[thread_id] = 0;
  __shared__ float valid[THREADS]; valid[thread_id] = 0;
  __shared__ float total[THREADS]; total[thread_id] = 0;

  __syncthreads();

  for (int n=0; n<1; n++) {

    if (thread_id < 3) {
      ti[thread_id] = poses[ix][thread_id];
      tj[thread_id] = poses[jx][thread_id];
    }

    if (thread_id < 4) {
      qi[thread_id] = poses[ix][thread_id+3];
      qj[thread_id] = poses[jx][thread_id+3];
    }

    __syncthreads();


    relSE3(ti, qi, tj, qj, tij, qij);

    float d, du, dv;
    float2 xnyn;

    GPU_1D_KERNEL_LOOP(k, ht*wd) {
      const int i = k / wd;
      const int j = k % wd;

      const float u = static_cast<float>(j);
      const float v = static_cast<float>(i);


      // if (disps[ix][i][j] < 0.01) {
      //   continue;
      // }
      
      // homogenous coordinates
      iproj(u, v, intrinsics_, Xi, disps[ix][i][j]);

      // transform homogenous point
      actSE3(tij, qij, Xi, Xj);

      xnyn = proj(Xj, intrinsics_);
      du = xnyn.x - u;
      dv = xnyn.y - v;
      d = sqrtf(du*du + dv*dv);

      total[threadIdx.x] += beta;
      
      if (Xj[2] > MIN_DEPTH) {
        accum[threadIdx.x] += beta * d;
        valid[threadIdx.x] += beta;
      }

      // homogenous coordinates
      iproj(u, v, intrinsics_, Xi, disps[ix][i][j]);

      Xj[0] = Xi[0] + Xi[3] * tij[0];
      Xj[1] = Xi[1] + Xi[3] * tij[1];
      Xj[2] = Xi[2] + Xi[3] * tij[2];

      xnyn = proj(Xj, intrinsics_);
      du = xnyn.x - u;
      dv = xnyn.y - v;
      d = sqrtf(du*du + dv*dv);

      total[threadIdx.x] += (1 - beta);
      
      if (Xj[2] > MIN_DEPTH) {
        accum[threadIdx.x] += (1 - beta) * d;
        valid[threadIdx.x] += (1 - beta);
      }
    }

    if (threadIdx.x == 0) {
      int tmp = ix;
      ix = jx;
      jx = tmp;
    }

    __syncthreads();

  }
  __syncthreads(); blockReduce(accum);
  __syncthreads(); blockReduce(total);
  __syncthreads(); blockReduce(valid);

  __syncthreads();

  if (thread_id == 0) {
    dist[block_id] = (valid[0] / (total[0] + 1e-8) < 0.75) ? 1000.0 : accum[0] / valid[0];
  }
}

__global__ void covis_distance_kernel(
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> poses,
    const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> disps,
    const torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> intrinsics,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> ii,
    torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> dist) {

  const int block_id = blockIdx.x;
  const int thread_id = threadIdx.x;

  const int ht = disps.size(1);
  const int wd = disps.size(2);

  __shared__ int ix;
  __shared__ int jx1, jx2;

  __shared__ float intrinsics_[4];

  __shared__ float ti[3], tj1[3], tij1[3], tj2[3], tij2[3];
  __shared__ float qi[4], qj1[4], qij1[4], qj2[4], qij2[4];

  // load intrinsics from global memory
  if (thread_id == 0) {
    ix = static_cast<int>(ii[block_id]);
    jx1 = ix - 1;
    jx2 = ix + 1;
    intrinsics_[0] = intrinsics[0];
    intrinsics_[1] = intrinsics[1];
    intrinsics_[2] = intrinsics[2];
    intrinsics_[3] = intrinsics[3];
  }

  __syncthreads();

  //points 
  float Xi[4];
  float Xj1[4], Xj2[4];

  __shared__ float accum[THREADS]; accum[thread_id] = 0;

  __syncthreads();

  for (int n=0; n<1; n++) {

    if (thread_id < 3) {
      ti[thread_id] = poses[ix][thread_id];
      tj1[thread_id] = poses[jx1][thread_id];
      tj2[thread_id] = poses[jx2][thread_id];
    }

    if (thread_id < 4) {
      qi[thread_id] = poses[ix][thread_id+3];
      qj1[thread_id] = poses[jx1][thread_id+3];
      qj2[thread_id] = poses[jx2][thread_id+3];
    }

    __syncthreads();

    relSE3(ti, qi, tj1, qj1, tij1, qij1);
    relSE3(ti, qi, tj2, qj2, tij2, qij2);

    float2 xnyn1, xnyn2;

    GPU_1D_KERNEL_LOOP(k, ht*wd) {
      const int i = k / wd;
      const int j = k % wd;

      const float u = static_cast<float>(j);
      const float v = static_cast<float>(i);

      // homogenous coordinates
      iproj(u, v, intrinsics_, Xi, disps[ix][i][j]);

      // transform homogenous point
      actSE3(tij1, qij1, Xi, Xj1);
      actSE3(tij2, qij2, Xi, Xj2);

      xnyn1 = proj(Xj1, intrinsics_);
      xnyn2 = proj(Xj2, intrinsics_);
      bool out1 = xnyn1.x < 0 || xnyn1.x >= wd || xnyn1.y < 0 || xnyn1.y >= ht;
      bool out2 = xnyn2.x < 0 || xnyn2.x >= wd || xnyn2.y < 0 || xnyn2.y >= ht;
      if (out1 && out2) {
        accum[threadIdx.x] += 1;
      }
    }

    __syncthreads();
  }
  __syncthreads(); blockReduce(accum);
  __syncthreads();

  if (thread_id == 0) {
    dist[block_id] = accum[0] / (ht*wd);
  }
}



__global__ void depth_filter_kernel(
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> poses,
    const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> disps,
    const torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> intrinsics,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> inds,
    const torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> thresh,
    torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> counter)
{

  const int block_id = blockIdx.x;
  const int neigh_id = blockIdx.y;
  const int index = blockIdx.z * blockDim.x + threadIdx.x;

  // if (threadIdx.x == 0) {
  //   printf("%d %d %d %d\n", blockIdx.x, blockIdx.y, blockDim.x, threadIdx.x);
  // }

  const int num = disps.size(0);
  const int ht = disps.size(1);
  const int wd = disps.size(2);

  __shared__ int ix;
  __shared__ int jx;

  __shared__ float intrinsics_[4];

  __shared__ float ti[3], tj[3], tij[3];
  __shared__ float qi[4], qj[4], qij[4];

  if (threadIdx.x == 0) {
    ix = static_cast<int>(inds[block_id]);
    jx = (neigh_id < 3) ? ix - neigh_id - 1 : ix + neigh_id;
    intrinsics_[0] = intrinsics[0];
    intrinsics_[1] = intrinsics[1];
    intrinsics_[2] = intrinsics[2];
    intrinsics_[3] = intrinsics[3];
  }

  __syncthreads();

  if (jx < 0 || jx >= num) {
    return;
  }

  const float t = thresh[block_id];

  // load poses from global memory
  if (threadIdx.x < 3) {
    ti[threadIdx.x] = poses[ix][threadIdx.x];
    tj[threadIdx.x] = poses[jx][threadIdx.x];
  }

  if (threadIdx.x < 4) {
    qi[threadIdx.x] = poses[ix][threadIdx.x+3];
    qj[threadIdx.x] = poses[jx][threadIdx.x+3];
  }

  __syncthreads();

  if (threadIdx.x == 0) {
    relSE3(ti, qi, tj, qj, tij, qij);
  }

  //points 
  float Xi[4];
  float Xj[4];
  float2 xnyn;

  __syncthreads();

  if (index < ht*wd) {
    const int i = index / wd;
    const int j = index % wd;

    const float ui = static_cast<float>(j);
    const float vi = static_cast<float>(i);
    const float di = disps[ix][i][j];
    
    // homogenous coordinates
    iproj(ui, vi, intrinsics_, Xi, di);

    // transform homogenous point
    actSE3(tij, qij, Xi, Xj);

    xnyn = proj(Xj, intrinsics_);
    const float uj = xnyn.x;
    const float vj = xnyn.y;
    const float dj = Xj[3] / Xj[2];

    const int u0 = static_cast<int>(floor(uj));
    const int v0 = static_cast<int>(floor(vj));

    if (u0 >= 0 && v0 >= 0 && u0 < wd-1 && v0 < ht-1) {
      const float wx = ceil(uj) - uj;
      const float wy = ceil(vj) - vj;

      const float d00 = disps[jx][v0+0][u0+0];
      const float d01 = disps[jx][v0+0][u0+1];
      const float d10 = disps[jx][v0+1][u0+0];
      const float d11 = disps[jx][v0+1][u0+1];

      const float dj_hat = wy*wx*d00 + wy*(1-wx)*d01 + (1-wy)*wx*d10 + (1-wy)*(1-wx)*d11;

      const float err = abs(1.0/dj - 1.0/dj_hat);
      if       (abs(1.0/dj - 1.0/d00) < t) atomicAdd(&counter[block_id][i][j], 1.0f);
      else if  (abs(1.0/dj - 1.0/d01) < t) atomicAdd(&counter[block_id][i][j], 1.0f);
      else if  (abs(1.0/dj - 1.0/d10) < t) atomicAdd(&counter[block_id][i][j], 1.0f);
      else if  (abs(1.0/dj - 1.0/d11) < t) atomicAdd(&counter[block_id][i][j], 1.0f);
    }
  }
}



__global__ void iproj_kernel(
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> poses,
    const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> disps,
    const torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> intrinsics,
    torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> points)

{

  const int block_id = blockIdx.x;
  const int index = blockIdx.y * blockDim.x + threadIdx.x;


  const int num = disps.size(0);
  const int ht = disps.size(1);
  const int wd = disps.size(2);

  __shared__ float intrinsics_[4];

  __shared__ float t[3];
  __shared__ float q[4];

  if (threadIdx.x == 0) {
    intrinsics_[0] = intrinsics[0];
    intrinsics_[1] = intrinsics[1];
    intrinsics_[2] = intrinsics[2];
    intrinsics_[3] = intrinsics[3];
  }

  __syncthreads();


  // load poses from global memory
  if (threadIdx.x < 3) {
    t[threadIdx.x] = poses[block_id][threadIdx.x];
  }

  if (threadIdx.x < 4) {
    q[threadIdx.x] = poses[block_id][threadIdx.x+3];
  }

  __syncthreads();

  //points 
  float Xi[4];
  float Xj[4];

  if (index < ht*wd) {
    const int i = index / wd;
    const int j = index % wd;

    const float ui = static_cast<float>(j);
    const float vi = static_cast<float>(i);
    const float di = disps[block_id][i][j];
    
    // homogenous coordinates
    iproj(ui, vi, intrinsics_, Xi, di);

    // transform homogenous point
    actSE3(t, q, Xi, Xj);

    points[block_id][i][j][0] = Xj[0] / Xj[3];
    points[block_id][i][j][1] = Xj[1] / Xj[3];
    points[block_id][i][j][2] = Xj[2] / Xj[3];

  }
}

__global__ void bi_inter_kernel(
    const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> scales,
    const torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> grids,
    torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> outputs,
    torch::PackedTensorAccessor32<float,5,torch::RestrictPtrTraits> Jacobis)

{

  const int block_id = blockIdx.x;
  const int index = blockIdx.y * blockDim.x + threadIdx.x;

  const int num = grids.size(0);
  const int ht = grids.size(1);
  const int wd = grids.size(2);

  __syncthreads();

  if (index < ht*wd) {
    const int i = index / wd;
    const int j = index % wd;

    const float xind = grids[block_id][i][j][0];
    const float yind = grids[block_id][i][j][1];

    int x0 = floor(xind);
    int y0 = floor(yind);
    int x1 = x0 + 1;
    int y1 = y0 + 1;

    float wa = (x1 - xind) * (y1 - yind);
    float wb = (x1 - xind) * (yind - y0);
    float wc = (xind - x0) * (y1 - yind);
    float wd = (xind - x0) * (yind - y0);

    outputs[block_id][i][j] = wa * scales[block_id][y0][x0] + wb * scales[block_id][y1][x0] + wc * scales[block_id][y0][x1] + wd * scales[block_id][y1][x1];
    Jacobis[block_id][i][j][y0][x0] = wa;
    Jacobis[block_id][i][j][y1][x0] = wb;
    Jacobis[block_id][i][j][y0][x1] = wc;
    Jacobis[block_id][i][j][y1][x1] = wd;
  }
}



__global__ void accum_kernel(
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> inps,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> ptrs,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> idxs,
    torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> outs)
{
  
  const int block_id = blockIdx.x;
  const int D = inps.size(2);

  const int start = ptrs[block_id];
  const int end = ptrs[block_id+1];

  for (int k=threadIdx.x; k<D; k+=blockDim.x) {
    float x = 0;
    for (int i=start; i<end; i++) {
      x += inps[idxs[i]][k];
    }
    outs[block_id][k] = x;
  }  
}


__device__ void
retrSE3(const float *xi, const float* t, const float* q, float* t1, float* q1) {
  // retraction on SE3 manifold

  float dt[3] = {0, 0, 0};
  float dq[4] = {0, 0, 0, 1};
  
  expSE3(xi, dt, dq);

  q1[0] = dq[3] * q[0] + dq[0] * q[3] + dq[1] * q[2] - dq[2] * q[1];
  q1[1] = dq[3] * q[1] + dq[1] * q[3] + dq[2] * q[0] - dq[0] * q[2];
  q1[2] = dq[3] * q[2] + dq[2] * q[3] + dq[0] * q[1] - dq[1] * q[0];
  q1[3] = dq[3] * q[3] - dq[0] * q[0] - dq[1] * q[1] - dq[2] * q[2];

  actSO3(dq, t, t1);
  t1[0] += dt[0];
  t1[1] += dt[1];
  t1[2] += dt[2];
}


__global__ void pose_retr_kernel(
    torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> poses,
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> dx,
    const int t0, const int t1) 
{

  for (int k=t0+threadIdx.x; k<t1; k+=blockDim.x) {
    float xi[6], q[4], q1[4], t[3], t1[3];

    t[0] = poses[k][0];
    t[1] = poses[k][1];
    t[2] = poses[k][2];

    q[0] = poses[k][3];
    q[1] = poses[k][4];
    q[2] = poses[k][5];
    q[3] = poses[k][6];
    
    for (int n=0; n<6; n++) {
      xi[n] = dx[k-t0][n];
    }

    retrSE3(xi, t, q, t1, q1);

    poses[k][0] = t1[0];
    poses[k][1] = t1[1];
    poses[k][2] = t1[2];

    poses[k][3] = q1[0];
    poses[k][4] = q1[1];
    poses[k][5] = q1[2];
    poses[k][6] = q1[3];
  }
}

__global__ void disp_retr_kernel(
    torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> disps,
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> dz,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> inds) 
{
  const int i = inds[blockIdx.x];
  const int ht = disps.size(1);
  const int wd = disps.size(2);

  for (int k=threadIdx.x; k<ht*wd; k+=blockDim.x) {
    float d = disps[i][k/wd][k%wd] + dz[blockIdx.x][k];
    disps[i][k/wd][k%wd] = d;
  }
}

torch::Tensor accum_cuda(torch::Tensor data, torch::Tensor ix, torch::Tensor jx) {
  torch::Tensor ix_cpu = ix.to(torch::kCPU);
  torch::Tensor jx_cpu = jx.to(torch::kCPU);
  torch::Tensor inds = torch::argsort(ix_cpu);

  long* ix_data = ix_cpu.data_ptr<long>();
  long* jx_data = jx_cpu.data_ptr<long>();
  long* kx_data = inds.data_ptr<long>();

  int count = jx.size(0);
  std::vector<int> cols;

  torch::Tensor ptrs_cpu = torch::zeros({count+1}, 
    torch::TensorOptions().dtype(torch::kInt64));
  
  long* ptrs_data = ptrs_cpu.data_ptr<long>();
  ptrs_data[0] = 0;

  int i = 0;
  for (int j=0; j<count; j++) {
    while (i < ix.size(0) && ix_data[kx_data[i]] <= jx_data[j]) {
      if (ix_data[kx_data[i]] == jx_data[j])
        cols.push_back(kx_data[i]);
      i++;
    }
    ptrs_data[j+1] = cols.size();
  }

  torch::Tensor idxs_cpu = torch::zeros({long(cols.size())}, 
    torch::TensorOptions().dtype(torch::kInt64));

  long* idxs_data = idxs_cpu.data_ptr<long>();

  for (int i=0; i<cols.size(); i++) {
    idxs_data[i] = cols[i];
  }

  torch::Tensor ptrs = ptrs_cpu.to(torch::kCUDA);
  torch::Tensor idxs = idxs_cpu.to(torch::kCUDA);

  torch::Tensor out = torch::zeros({jx.size(0), data.size(1)},
    torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA));

  accum_kernel<<<count, THREADS>>>(
    data.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    ptrs.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
    idxs.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
    out.packed_accessor32<float,2,torch::RestrictPtrTraits>());

  return out;
}


__global__ void EEt6x6_kernel(
    const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> E,
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> Q,
    const torch::PackedTensorAccessor32<long,2,torch::RestrictPtrTraits> idx,
    torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> S)
{

  // indicices
  const int ix = idx[blockIdx.x][0];
  const int jx = idx[blockIdx.x][1];
  const int kx = idx[blockIdx.x][2];

  const int d = E.size(1);
  const int D = E.size(2);

  float dS[7][7];
  float ei[7];
  float ej[7];

  for (int i=0; i<d; i++) {
    for (int j=0; j<d; j++) {
      dS[i][j] = 0;
    }
  }

  for (int k=threadIdx.x; k<D; k+=blockDim.x) {
    const float q = Q[kx][k];
      
    // coalesced memory read
    for (int n=0; n<d; n++) {
      ei[n] = E[ix][n][k] * q;
      ej[n] = E[jx][n][k];
    }

    // block EEt
    for (int n=0; n<d; n++) {
      for (int m=0; m<d; m++) {
        dS[n][m] += ei[n] * ej[m];
      }
    }
  }

  __syncthreads();
  __shared__ float sdata[THREADS];

  for (int n=0; n<d; n++) {
    for (int m=0; m<d; m++) {
      sdata[threadIdx.x] = dS[n][m];

      blockReduce(sdata);

      if (threadIdx.x == 0) {
        S[blockIdx.x][n][m] = sdata[0];
      }
    }
  }
}


__global__ void Ev6x1_kernel(
    const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> E,
    const torch::PackedTensorAccessor32<float, 2,torch::RestrictPtrTraits> Q,
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> w,
    const torch::PackedTensorAccessor32<long,2,torch::RestrictPtrTraits> idx,
    torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> v)
{
  const int d = E.size(1);
  const int D = E.size(2);
  const int kx = idx[blockIdx.x][0];

  float b[7];
  for (int n=0; n<d; n++) {
    b[n] = 0.0;
  }

  for (int k=threadIdx.x; k<D; k+=blockDim.x) {
    const float q_w = Q[kx][k] * w[kx][k];

    for (int n=0; n<d; n++) {
      b[n] += q_w * E[blockIdx.x][n][k];
    }
  }

  __syncthreads();
  __shared__ float sdata[THREADS];

  for (int n=0; n<d; n++) {
    sdata[threadIdx.x] = b[n];
    blockReduce(sdata);

    if (threadIdx.x == 0) {
      v[blockIdx.x][n] += sdata[0];
    }
  }
}

__global__ void EvT6x1_kernel(
  const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> E,
  const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> x,
  const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> idx,
  torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> w)
{
  const int d = E.size(1);
  const int D = E.size(2);
  const int ix = idx[blockIdx.x];

  if (idx[blockIdx.x] <= 0 || idx[blockIdx.x] >= x.size(0))
    return;

  for (int k=threadIdx.x; k<D; k+=blockDim.x) {
    float dw = 0;
    for (int n=0; n<d; n++) {
      dw += E[blockIdx.x][n][k] * x[ix][n];
    }
    w[blockIdx.x][k] = dw;
  }
}

class SparseBlock {
  public:

    Eigen::SparseMatrix<double> A;
    Eigen::VectorX<double> b;

    SparseBlock(int N, int M) : N(N), M(M) {
      A = Eigen::SparseMatrix<double>(N*M, N*M);
      b = Eigen::VectorXd::Zero(N*M);
    }

    SparseBlock(Eigen::SparseMatrix<double> const& A, Eigen::VectorX<double> const& b, 
        int N, int M) : A(A), b(b), N(N), M(M) {}

    void update_lhs(torch::Tensor As, torch::Tensor ii, torch::Tensor jj) {

      auto As_cpu = As.to(torch::kCPU).to(torch::kFloat64);
      auto ii_cpu = ii.to(torch::kCPU).to(torch::kInt64);
      auto jj_cpu = jj.to(torch::kCPU).to(torch::kInt64);

      auto As_acc = As_cpu.accessor<double,3>();
      auto ii_acc = ii_cpu.accessor<long,1>();
      auto jj_acc = jj_cpu.accessor<long,1>();

      std::vector<T> tripletList;
      for (int n=0; n<ii.size(0); n++) {
        const int i = ii_acc[n];
        const int j = jj_acc[n];

        if (i >= 0 && j >= 0) {
          for (int k=0; k<M; k++) {
            for (int l=0; l<M; l++) {
              double val = As_acc[n][k][l];
              tripletList.push_back(T(M*i + k, M*j + l, val));
            }
          }
        }
      }
      A.setFromTriplets(tripletList.begin(), tripletList.end());
    }

    void update_rhs(torch::Tensor bs, torch::Tensor ii) {
      auto bs_cpu = bs.to(torch::kCPU).to(torch::kFloat64);
      auto ii_cpu = ii.to(torch::kCPU).to(torch::kInt64);

      auto bs_acc = bs_cpu.accessor<double,2>();
      auto ii_acc = ii_cpu.accessor<long,1>();

      for (int n=0; n<ii.size(0); n++) {
        const int i = ii_acc[n];
        if (i >= 0) {
          for (int j=0; j<M; j++) {
            b(i*M + j) += bs_acc[n][j];
          }
        }
      }
    }

    SparseBlock operator-(const SparseBlock& S) {
      return SparseBlock(A - S.A, b - S.b, N, M);
    }

    std::tuple<torch::Tensor, torch::Tensor> get_dense() {
      Eigen::MatrixXd Ad = Eigen::MatrixXd(A);

      torch::Tensor H = torch::from_blob(Ad.data(), {N*M, N*M}, torch::TensorOptions()
        .dtype(torch::kFloat64)).to(torch::kCUDA).to(torch::kFloat32);

      torch::Tensor v = torch::from_blob(b.data(), {N*M, 1}, torch::TensorOptions()
        .dtype(torch::kFloat64)).to(torch::kCUDA).to(torch::kFloat32);

      return std::make_tuple(H, v);

    }

    torch::Tensor solve(const float lm=0.0001, const float ep=0.1) {

      torch::Tensor dx;

      Eigen::SparseMatrix<double> L(A);
      L.diagonal().array() += ep + lm * L.diagonal().array();

      Eigen::SimplicialLLT<Eigen::SparseMatrix<double>> solver;
      solver.compute(L);

      if (solver.info() == Eigen::Success) {
        Eigen::VectorXd x = solver.solve(b);
        dx = torch::from_blob(x.data(), {N, M}, torch::TensorOptions()
          .dtype(torch::kFloat64)).to(torch::kCUDA).to(torch::kFloat32);
      }
      else {
        dx = torch::zeros({N, M}, torch::TensorOptions()
          .device(torch::kCUDA).dtype(torch::kFloat32));
      }
      
      return dx;
    }

  private:
    const int N;
    const int M;

};


SparseBlock schur_block(torch::Tensor E,
                        torch::Tensor Q,
                        torch::Tensor w,
                        torch::Tensor ii,
                        torch::Tensor jj,
                        torch::Tensor kk,
                        const int t0,
                        const int t1)
{

  torch::Tensor ii_cpu = ii.to(torch::kCPU);
  torch::Tensor jj_cpu = jj.to(torch::kCPU);
  torch::Tensor kk_cpu = kk.to(torch::kCPU);

  const int D = E.size(1);
  const int P = t1 - t0;
  const long* ii_data = ii_cpu.data_ptr<long>();
  const long* jj_data = jj_cpu.data_ptr<long>();
  const long* kk_data = kk_cpu.data_ptr<long>();

  std::vector<std::vector<long>> graph(P);
  std::vector<std::vector<long>> index(P);

  for (int n=0; n<ii_cpu.size(0); n++) {
    const int j = jj_data[n];
    const int k = kk_data[n];

    if (j >= t0 && j <= t1) {
      const int t = j - t0;
      graph[t].push_back(k);
      index[t].push_back(n);
    }
  }

  std::vector<long> ii_list, jj_list, idx, jdx;

  for (int i=0; i<P; i++) {
    for (int j=0; j<P; j++) {
      for (int k=0; k < graph[i].size(); k++) {
        for (int l=0; l < graph[j].size(); l++) {
          if (graph[i][k] == graph[j][l]) {
            ii_list.push_back(i);
            jj_list.push_back(j);

            idx.push_back(index[i][k]);
            idx.push_back(index[j][l]);
            idx.push_back(graph[i][k]);
          }
        }
      }
    }
  }

  torch::Tensor ix_cuda = torch::from_blob(idx.data(), {long(idx.size())}, 
    torch::TensorOptions().dtype(torch::kInt64)).to(torch::kCUDA).view({-1, 3});

  torch::Tensor jx_cuda = torch::stack({kk_cpu}, -1)
    .to(torch::kCUDA).to(torch::kInt64);

  torch::Tensor ii2_cpu = torch::from_blob(ii_list.data(), {long(ii_list.size())}, 
    torch::TensorOptions().dtype(torch::kInt64)).view({-1});

  torch::Tensor jj2_cpu = torch::from_blob(jj_list.data(), {long(jj_list.size())}, 
    torch::TensorOptions().dtype(torch::kInt64)).view({-1});

  torch::Tensor S = torch::zeros({ix_cuda.size(0), D, D}, 
    torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA));

  torch::Tensor v = torch::zeros({jx_cuda.size(0), D},
    torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA));

  EEt6x6_kernel<<<ix_cuda.size(0), THREADS>>>(
    E.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    Q.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    ix_cuda.packed_accessor32<long,2,torch::RestrictPtrTraits>(),
    S.packed_accessor32<float,3,torch::RestrictPtrTraits>());

  Ev6x1_kernel<<<jx_cuda.size(0), THREADS>>>(
    E.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    Q.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    w.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    jx_cuda.packed_accessor32<long,2,torch::RestrictPtrTraits>(),
    v.packed_accessor32<float,2,torch::RestrictPtrTraits>());

  // schur block
  SparseBlock A(P, D);
  A.update_lhs(S, ii2_cpu, jj2_cpu);
  A.update_rhs(v, jj_cpu - t0);

  return A;
}

std::vector<torch::Tensor> proj_trans_cuda(
    torch::Tensor poses,
    torch::Tensor disps,
    torch::Tensor intrinsics,
    torch::Tensor targets,
    torch::Tensor weights,
    torch::Tensor ii,
    torch::Tensor jj)
{
  auto opts = poses.options();
  const int num = ii.size(0);
  const int ht = disps.size(1);
  const int wd = disps.size(2);

  std::tuple<torch::Tensor, torch::Tensor> kuniq = torch::_unique(ii, true, true);
  torch::Tensor kx = std::get<0>(kuniq);

  // initialize buffers
  torch::Tensor Cii = torch::zeros({num, ht*wd}, opts);
  torch::Tensor wi = torch::zeros({num, ht*wd}, opts);

  projective_transform2_kernel<<<num, THREADS>>>(
    targets.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
    weights.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
    poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    disps.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    intrinsics.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    ii.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
    jj.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
    Cii.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    wi.packed_accessor32<float,2,torch::RestrictPtrTraits>());

  torch::Tensor C = accum_cuda(Cii, ii, kx);
  torch::Tensor w = accum_cuda(wi, ii, kx);

  return {C, w};
}


std::vector<torch::Tensor> ba_cuda(
    torch::Tensor poses,
    torch::Tensor disps,
    torch::Tensor intrinsics,
    torch::Tensor targets,
    torch::Tensor weights,
    torch::Tensor eta,
    torch::Tensor ii,
    torch::Tensor jj,
    const int t0,
    const int t1,
    const int iterations,
    const float lm,
    const float ep,
    const bool motion_only)
{
  auto opts = poses.options();
  const int num = ii.size(0);
  const int ht = disps.size(1);
  const int wd = disps.size(2);

  torch::Tensor ts = torch::arange(t0, t1).to(torch::kCUDA);
  torch::Tensor ii_exp = torch::cat({ts, ii}, 0);
  torch::Tensor jj_exp = torch::cat({ts, jj}, 0);

  std::tuple<torch::Tensor, torch::Tensor> kuniq = 
    torch::_unique(ii_exp, true, true);

  torch::Tensor kx = std::get<0>(kuniq);
  torch::Tensor kk_exp = std::get<1>(kuniq);

  torch::Tensor dx;
  torch::Tensor dz;
  torch::Tensor Linv, dzcov;

  // initialize buffers
  torch::Tensor Hs = torch::zeros({4, num, 6, 6}, opts);
  torch::Tensor vs = torch::zeros({2, num, 6}, opts);
  torch::Tensor Eii = torch::zeros({num, 6, ht*wd}, opts);
  torch::Tensor Eij = torch::zeros({num, 6, ht*wd}, opts);
  torch::Tensor Cii = torch::zeros({num, ht*wd}, opts);
  torch::Tensor wi = torch::zeros({num, ht*wd}, opts);

  for (int itr=0; itr<iterations; itr++) {

    projective_transform_kernel<<<num, THREADS>>>(
      targets.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
      weights.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
      poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
      disps.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
      intrinsics.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
      ii.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
      jj.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
      Hs.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
      vs.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
      Eii.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
      Eij.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
      Cii.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
      wi.packed_accessor32<float,2,torch::RestrictPtrTraits>());


    // pose x pose block
    SparseBlock A(t1 - t0, 6);

    A.update_lhs(Hs.reshape({-1, 6, 6}), 
        torch::cat({ii, ii, jj, jj}) - t0, 
        torch::cat({ii, jj, ii, jj}) - t0);

    A.update_rhs(vs.reshape({-1, 6}), 
        torch::cat({ii, jj}) - t0);

    if (motion_only) {
      dx = A.solve(lm, ep);

      // update poses
      pose_retr_kernel<<<1, THREADS>>>(
        poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
        dx.packed_accessor32<float,2,torch::RestrictPtrTraits>(), t0, t1);
    }
    
    else {
      torch::Tensor C = accum_cuda(Cii, ii, kx);
      torch::Tensor w = accum_cuda(wi, ii, kx);
      torch::Tensor Q = 1.0 / (C + eta.view({-1, ht*wd}));

      torch::Tensor Ei = accum_cuda(Eii.view({num, 6*ht*wd}), ii, ts).view({t1-t0, 6, ht*wd});
      torch::Tensor E = torch::cat({Ei, Eij}, 0);

      SparseBlock S = schur_block(E, Q, w, ii_exp, jj_exp, kk_exp, t0, t1);
      dx = (A - S).solve(lm, ep);

      torch::Tensor ix = jj_exp - t0;
      torch::Tensor dw = torch::zeros({ix.size(0), ht*wd}, opts);

      EvT6x1_kernel<<<ix.size(0), THREADS>>>(
        E.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
        dx.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
        ix.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
        dw.packed_accessor32<float,2,torch::RestrictPtrTraits>());

      dz = Q * (w - accum_cuda(dw, ii_exp, kx));

      // update poses
      pose_retr_kernel<<<1, THREADS>>>(
        poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
        dx.packed_accessor32<float,2,torch::RestrictPtrTraits>(), t0, t1);

      // update disparity maps
      disp_retr_kernel<<<kx.size(0), THREADS>>>(
        disps.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
        dz.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
        kx.packed_accessor32<long,1,torch::RestrictPtrTraits>());
    }

  }

  return {dx, dz, dzcov};
}


std::vector<torch::Tensor> pgba_cuda(
    torch::Tensor poses,
    torch::Tensor disps,
    torch::Tensor eta,
    torch::Tensor Hs,
    torch::Tensor vs,
    torch::Tensor Eii,
    torch::Tensor Eij,
    torch::Tensor Cii,
    torch::Tensor wi,
    torch::Tensor Hsp,
    torch::Tensor vsp,
    torch::Tensor ii,
    torch::Tensor jj,
    torch::Tensor iip,
    torch::Tensor jjp,
    const int t0,
    const int t1,
    const float lm,
    const float ep)
{
  auto opts = poses.options();
  const int num = ii.size(0);
  const int ht = disps.size(1);
  const int wd = disps.size(2);
  const int D = 7;

  torch::Tensor ts = torch::arange(t0, t1).to(torch::kCUDA);
  torch::Tensor ii_exp = torch::cat({ts, ii}, 0);
  torch::Tensor jj_exp = torch::cat({ts, jj}, 0);

  std::tuple<torch::Tensor, torch::Tensor> kuniq = 
    torch::_unique(ii_exp, true, true);

  torch::Tensor kx = std::get<0>(kuniq);
  torch::Tensor kk_exp = std::get<1>(kuniq);

  torch::Tensor dx, dz;
  torch::Tensor Linv, dzcov;

  // pose x pose block
  SparseBlock A(t1 - t0, D);

  // add constraints
  torch::Tensor iii = torch::arange(0, t1-1).to(torch::kCUDA);
  torch::Tensor jji = iii + 1;
  torch::Tensor Hs_all = torch::cat({Hs.reshape({-1,D,D}), Hsp.reshape({-1,D,D})});
  torch::Tensor vs_all = torch::cat({vs.reshape({-1,D}), vsp.reshape({-1,D})});
  torch::Tensor ind1_all = torch::cat({ii, ii, jj, jj, iip, iip, jjp, jjp});
  torch::Tensor ind2_all = torch::cat({ii, jj, ii, jj, iip, jjp, iip, jjp});
  torch::Tensor ind3_all = torch::cat({ii, jj, iip, jjp});

  A.update_lhs(Hs_all, ind1_all - t0, ind2_all - t0);
  A.update_rhs(vs_all, ind3_all - t0);

  // solve system
  torch::Tensor C = accum_cuda(Cii, ii, kx);
  torch::Tensor w = accum_cuda(wi, ii, kx);
  torch::Tensor Q = 1.0 / (C + eta.view({-1, ht*wd}));

  torch::Tensor Ei = accum_cuda(Eii.view({num, D*ht*wd}), ii, ts).view({t1-t0, D, ht*wd});
  torch::Tensor E = torch::cat({Ei, Eij}, 0);

  SparseBlock S = schur_block(E, Q, w, ii_exp, jj_exp, kk_exp, t0, t1);
  SparseBlock B = A - S;
  dx = B.solve(lm, ep);
  torch::Tensor ix = jj_exp - t0;
  torch::Tensor dw = torch::zeros({ix.size(0), ht*wd}, opts);

  EvT6x1_kernel<<<ix.size(0), THREADS>>>(
    E.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    dx.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    ix.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
    dw.packed_accessor32<float,2,torch::RestrictPtrTraits>());

  dz = Q * (w - accum_cuda(dw, ii_exp, kx));

  // update disparity maps
  disp_retr_kernel<<<kx.size(0), THREADS>>>(
    disps.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    dz.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    kx.packed_accessor32<long,1,torch::RestrictPtrTraits>());

  return {dx, dz};
}


torch::Tensor frame_distance_cuda(
    torch::Tensor poses,
    torch::Tensor disps,
    torch::Tensor intrinsics,
    torch::Tensor ii,
    torch::Tensor jj,
    const float beta)
{
  auto opts = poses.options();
  const int num = ii.size(0);

  torch::Tensor dist = torch::zeros({num}, opts);

  frame_distance_kernel<<<num, THREADS>>>(
    poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    disps.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    intrinsics.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
    ii.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
    jj.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
    dist.packed_accessor32<float,1,torch::RestrictPtrTraits>(), beta);

  return dist;
}

torch::Tensor covis_distance_cuda(
    torch::Tensor poses,
    torch::Tensor disps,
    torch::Tensor intrinsics,
    torch::Tensor ii)
{
  auto opts = poses.options();
  const int num = ii.size(0);

  torch::Tensor dist = torch::zeros({num}, opts);

  covis_distance_kernel<<<num, THREADS>>>(
    poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    disps.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    intrinsics.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
    ii.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
    dist.packed_accessor32<float,1,torch::RestrictPtrTraits>());

  return dist;
}


torch::Tensor depth_filter_cuda(
    torch::Tensor poses,
    torch::Tensor disps,
    torch::Tensor intrinsics,
    torch::Tensor ix,
    torch::Tensor thresh)
{
  const int num = ix.size(0);
  const int ht = disps.size(1);
  const int wd = disps.size(2);

  torch::Tensor counter = torch::zeros({num, ht, wd}, disps.options());

  dim3 blocks(num, 6, NUM_BLOCKS(ht * wd));

  depth_filter_kernel<<<blocks, THREADS>>>(
    poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    disps.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    intrinsics.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
    ix.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
    thresh.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
    counter.packed_accessor32<float,3,torch::RestrictPtrTraits>());

  return counter;
}


torch::Tensor iproj_cuda(
    torch::Tensor poses,
    torch::Tensor disps,
    torch::Tensor intrinsics)
{

  const int nm = disps.size(0);
  const int ht = disps.size(1);
  const int wd = disps.size(2);

  auto opts = disps.options();
  torch::Tensor points = torch::zeros({nm, ht, wd, 3}, opts);

  dim3 blocks(nm, NUM_BLOCKS(ht * wd));

  iproj_kernel<<<blocks, THREADS>>>(
    poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    disps.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    intrinsics.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
    points.packed_accessor32<float,4,torch::RestrictPtrTraits>());

  return points;

}

std::vector<torch::Tensor> bi_inter_cuda(
    torch::Tensor scales,
    torch::Tensor grids)
{

  const int nm = grids.size(0);
  const int ht = grids.size(1);
  const int wd = grids.size(2);
  const int hs = scales.size(1);
  const int ws = scales.size(2);

  auto opts = grids.options();
  torch::Tensor outputs = torch::zeros({nm, ht, wd}, opts);
  torch::Tensor Jacobis = torch::zeros({nm, ht, wd, hs, ws}, opts);

  dim3 blocks(nm, NUM_BLOCKS(ht * wd));

  bi_inter_kernel<<<blocks, THREADS>>>(
    scales.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    grids.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
    outputs.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    Jacobis.packed_accessor32<float,5,torch::RestrictPtrTraits>());

  return {outputs, Jacobis};

}
